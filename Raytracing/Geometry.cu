#include "hip/hip_runtime.h"
#include "geometry.h"
bool __device__ Plane::intersect(const Ray& ray, IntersectionData& data)
{
	// intersect a ray with a XZ plane:
	// if the ray is pointing to the horizon, or "up", but the plane is below us,
	// of if the ray is pointing down, and the plane is above us, we have no intersection
	if ((ray.start.y > y && ray.dir.y > -1e-9) || (ray.start.y < y && ray.dir.y < 1e-9))
		return false;
	else {
		float yDiff = ray.dir.y;
		float wantYDiff = ray.start.y - this->y;
		float mult = wantYDiff / -yDiff;
		
		// if the distance to the intersection (mult) doesn't optimize our current distance, bail out:
		if (mult > data.dist) return false;
		
		Vector p = ray.start + ray.dir * mult;
		if (fabs(p.x) > limit || fabs(p.z) > limit) 
			return false;
		
		// calculate intersection:
		data.p = p;
		data.dist = mult;
		data.normal = Vector(0, 1, 0);
		data.dNdx = Vector(1, 0, 0);
		data.dNdy = Vector(0, 0, 1);
		data.u = data.p.x;
		data.v = data.p.z;
//		data.g = this;
		return true;
	}
}
bool __device__ Sphere::intersect(const Ray& ray, IntersectionData& info)
{
	// compute the sphere intersection using a quadratic equation:
	Vector H = ray.start - center;
	float A = ray.dir.lengthSqr();
	float B = 2 * dot(H, ray.dir);
	float C = H.lengthSqr() - R*R;
	float Dscr = B*B - 4*A*C;
	if (Dscr < 0) return false; // no solutions to the quadratic equation - then we don't have an intersection.
	float x1, x2;
	x1 = (-B + sqrt(Dscr)) / (2*A);
	x2 = (-B - sqrt(Dscr)) / (2*A);
	float sol = x2; // get the closer of the two solutions...
	if (sol < 0) sol = x1; // ... but if it's behind us, opt for the other one
	if (sol < 0) return false; // ... still behind? Then the whole sphere is behind us - no intersection.
	
	// if the distance to the intersection doesn't optimize our current distance, bail out:
	if (sol > info.dist) return false;
	
	info.dist = sol;
	info.p = ray.start + ray.dir * sol;
	info.normal = info.p - center; // generate the normal by getting the direction from the center to the ip
	info.normal.normalize();
	float angle = atan2(info.p.z - center.z, info.p.x - center.x);
	info.u = (PI + angle)/(2*PI);
	info.v = 1.0 - (PI/2 + asin((info.p.y - center.y)/R)) / PI;
	info.dNdx = Vector(cos(angle + PI/2), 0, sin(angle + PI/2));
	info.dNdy = info.dNdx ^ info.normal;
	return true;
}

inline __device__  bool Cube::intersectCubeSide(const Ray& ray, const Vector& center, IntersectionData& data)
{
	if (fabs(ray.dir.y) < 1e-9) return false;

	float halfSide = this->side * 0.5;
	bool found = false;
	for (int side = -1; side <= 1; side += 2) {
		float yDiff = ray.dir.y;
		float wantYDiff = ray.start.y - (center.y + side * halfSide);
		float mult = wantYDiff / -yDiff;
		if (mult < 0) continue;
		if (mult > data.dist) continue;
		Vector p = ray.start + ray.dir * mult;
		if (p.x < center.x - halfSide ||
			p.x > center.x + halfSide ||
			p.z < center.z - halfSide ||
			p.z > center.z + halfSide) continue;
		data.p = ray.start + ray.dir * mult;
		data.dist = mult;
		data.normal = Vector(0, side, 0);
		data.dNdx = Vector(1, 0, 0);
		data.dNdy = Vector(0, 0, side);
		data.u = data.p.x - center.x;
		data.v = data.p.z - center.z;
		found = true;	
	}
	return found;
}

bool __device__  Cube::intersect(const Ray& ray, IntersectionData& data)
{
	// check for intersection with the negative Y and positive Y sides
	bool found = intersectCubeSide(ray, center, data);
	
	// check for intersection with the negative X and positive X sides
	if (intersectCubeSide(project(ray, 1, 0, 2), project(center, 1, 0, 2), data)) {
		found = true;
		data.normal = unproject(data.normal, 1, 0, 2);
		data.p = unproject(data.p, 1, 0, 2);
	}

	// check for intersection with the negative Z and positive Z sides
	if (intersectCubeSide(project(ray, 0, 2, 1), project(center, 0, 2, 1), data)) {
		found = true;
		data.normal = unproject(data.normal, 0, 2, 1);
		data.p = unproject(data.p, 0, 2, 1);
	}
	return found;
}